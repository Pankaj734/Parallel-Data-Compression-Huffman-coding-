#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <limits.h>
#include <math.h> 
#include <hip/hip_runtime.h>
#include <algorithm>


#define BLOCK_SIZE 1024
__device__ unsigned int counter, counter_2;
//__device__ unsigned int flag;
__constant__ const unsigned int INTMAX = 2147483647;

// struct for dictionary
struct huffmanDictionary{
    unsigned char bitSequence[255];
    unsigned char bitSequenceLength;
};

// struct for node
struct huffmanNode{
    unsigned char letter;  // char to store
    unsigned int frequency;  // frequency of the char
    struct huffmanNode * left;  // left sub tree
    struct huffmanNode * right;  // right sub tree
};

struct huffmanDictionary huffmanDictionary[256];
struct huffmanNode * huffmanNode_head;
struct huffmanNode huffmanTreeNode[512];

#define DEBUG 1

__device__ int findIndex(unsigned int *freq, unsigned int size,unsigned int search){
    for(int i=0;i<size;i++){
        if(freq[i] == search){
            return i;
        }
    }
    return -1;
}
__global__ void findLeastFrequent(unsigned int *freq, unsigned int *min, int size, unsigned int threads, unsigned int* count, unsigned int *index){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    counter_2 = 0;
    __syncthreads();
    int ind;
    
    if(id<threads){
        
        while(1){
            min[counter_2] = INTMAX;
            
            atomicMin(&min[counter_2], freq[id]);
            // Need global barrier
            __syncthreads();
            
            ind = findIndex(freq, threads, min[counter_2]);
            index[counter_2] = ind;
            // Need global barrier
            __syncthreads();
            freq[ind] = INTMAX;
            
            if(id == 0) atomicInc(&counter_2, size);
            // Need global barrier
            __syncthreads();

            min[counter_2] = INTMAX;
            
            atomicMin(&min[counter_2], freq[id]);
            // Need global barrier
            __syncthreads();
            
            ind = findIndex(freq, threads, min[counter_2]);
            index[counter_2] = ind;
            // Need global barrier
            __syncthreads();
            freq[ind] = min[counter_2] + min[counter_2-1];
            
            if(id == 0) atomicInc(&counter_2, size);
            // Need global barrier
            __syncthreads();
            

            if(min[counter_2] == INTMAX || min[counter_2-1] == INTMAX){
                count[0] = counter_2;
                break;
            }
            
        }
    }
}

__global__ void searchSimilarIndex(unsigned int *index, unsigned int *resultIndex, unsigned int *cnt, int threads){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    
    __syncthreads();
    counter = 0;
    if(id != threads){
        if(index[id] == index[threads]){
            
            int temp = atomicInc(&counter, threads+1);
            resultIndex[temp] = id;
        }
        __syncthreads();
        cnt[0] = counter;
    }
}



void buildHuffmanTree(int count,unsigned char *uniqueChar, unsigned int *frequency,int newIndex, int childIndex){
    if(count == 0){
        
        huffmanTreeNode[newIndex].frequency = frequency[childIndex];
        huffmanTreeNode[newIndex].letter = uniqueChar[childIndex];
        huffmanTreeNode[newIndex].left = NULL;
        huffmanTreeNode[newIndex].right = NULL;
    }
    else{
        
        huffmanTreeNode[newIndex].frequency = huffmanTreeNode[childIndex].frequency + huffmanTreeNode[childIndex + 1].frequency;
        huffmanTreeNode[newIndex].left = & huffmanTreeNode[childIndex];
        huffmanTreeNode[newIndex].right = & huffmanTreeNode[childIndex + 1];
        huffmanNode_head = & (huffmanTreeNode[newIndex]);
    }
}

void buildHuffmanDictionary(struct huffmanNode * root, unsigned char * bitSequence, unsigned char bitSequenceLength){
    if(root -> left){
        bitSequence[bitSequenceLength] = 0;
        buildHuffmanDictionary(root -> left, bitSequence, bitSequenceLength + 1);
    }

    if(root -> right){
        bitSequence[bitSequenceLength] = 1;
        buildHuffmanDictionary(root -> right, bitSequence, bitSequenceLength + 1);
    }

    // copy the bit sequence and the length to the dictionary
    if(root -> right == NULL && root -> left == NULL){
        huffmanDictionary[root -> letter].bitSequenceLength = bitSequenceLength;
        memcpy(huffmanDictionary[root -> letter].bitSequence, bitSequence, bitSequenceLength * sizeof(unsigned char));
    }
}


int main(int argc, char ** argv){

    clock_t start, end;
    unsigned int cpuTime;
    unsigned int compressedFileLength, outputFileLengthCounter, outputFileLength, extra;
    unsigned int distinctCharacterCount;
    unsigned int frequency[256];

    unsigned char bitSequenceLength = 0, bitSequence[255];
    unsigned char * compressedData, * outputData;
    struct huffmanNode * huffmanNode_current;

    FILE *compressedFile, *outputFile;

    // read input file get length, frequency and data
    compressedFile = fopen(argv[1], "r");
    
    fread(& outputFileLength, sizeof(unsigned int), 1, compressedFile);
    //no. of extra bits added, calculate here
    fread(& extra, sizeof(unsigned int), 1, compressedFile);
    fread(frequency, 256 * sizeof(unsigned int), 1, compressedFile);

    // find length of the compressed file
    fseek(compressedFile, 0, SEEK_END);
    compressedFileLength = ftell(compressedFile) - 1032;
    
    fseek(compressedFile, 1032, SEEK_SET);

    // allocate the required memory, read the file
    compressedData = (unsigned char *)malloc((compressedFileLength) * sizeof(unsigned char));
    fread(compressedData, sizeof(unsigned char), compressedFileLength, compressedFile);

    // start the clock, tick tick
    start = clock();

    // for(int i=0;i<256;i++) printf("%c ",frequency[i]);
    // printf("\n");
    // initialize the huffman tree
    distinctCharacterCount = 0;
    for(int i = 0; i < 256; i++){
        if(frequency[i] > 0){
            distinctCharacterCount ++;
        }
    }

    int unique = 0;
    unsigned char *uniqueChar, *duniqueChar;
    uniqueChar = (unsigned char *)malloc(256*sizeof(unsigned char));
    hipMalloc(&duniqueChar, 256*sizeof(unsigned char));
    for(int i = 0; i<256; i++){
        if(frequency[i] > 0){
            uniqueChar[unique++] = i;
            //printf("%d ",frequency[i]);
        }
    }
    //printf("\n");
    hipMemcpy(duniqueChar, uniqueChar, 256*sizeof(unsigned char), hipMemcpyHostToDevice);

    // *** FIND MINIMUM 2 FREQUENCY FOR ADDING NEW NODE ***
    unsigned int *tempFreq, *tempDFreq;
    unsigned int *min, *dmin;
    unsigned int *cntMin, *dcntMin;
    unsigned int *indMin, *dindMin;
    int ctr;

    tempFreq = (unsigned int *)malloc(unique*sizeof(unsigned int));
    
    min = (unsigned int *)malloc(outputFileLength*sizeof(unsigned int));
    cntMin = (unsigned int *)malloc(sizeof(unsigned int));
    indMin = (unsigned int *)malloc(outputFileLength*sizeof(unsigned int));
    ctr = 0;
    for(unsigned int i=0;i<256;i++){
        if(frequency[i]!=0){
            tempFreq[ctr++] = frequency[i];
        }
    }
    // for(unsigned int i=0;i<unique;i++) printf("%d:%c ",tempFreq[i],uniqueChar[i]);
    // printf("\n");
    
    hipMalloc(&tempDFreq, unique*sizeof(unsigned int));
    
    hipMalloc(&dmin, outputFileLength*sizeof(unsigned int));
    hipMalloc(&dindMin, outputFileLength*sizeof(unsigned int));
    hipMalloc(&dcntMin, sizeof(unsigned int));
    hipMemcpy(tempDFreq, tempFreq, unique*sizeof(unsigned int), hipMemcpyHostToDevice);

    float num = (float)(unique)/(float)BLOCK_SIZE;
    //num = (float)(5)/(float)BLOCK_SIZE;
    int mod = BLOCK_SIZE;
    if(unique < BLOCK_SIZE) mod = unique%BLOCK_SIZE;
    //if(5 < BLOCKSIZE) mod = 5%BLOCK_SIZE;
    int n = ceil(num);
    //printf("%d %d\n",n,mod);
    findLeastFrequent<<<n, mod>>>(tempDFreq, dmin, outputFileLength, unique, dcntMin, dindMin);
    
    hipDeviceSynchronize();

    hipMemcpy(min, dmin, outputFileLength*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(indMin, dindMin, outputFileLength*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(cntMin, dcntMin, sizeof(unsigned int), hipMemcpyDeviceToHost);
    // printf("count : %d\n",cntMin[0]);
    // for(unsigned int i=0;i<cntMin[0];i++){
    //     printf("%d:%d:%d ",i,indMin[i],min[i]);
    // } 
    // printf("\n");
    // printf("Min:\n");
    // for(unsigned int i=0;i<cntMin[0];i++) printf("%d ",min[i]);
    // printf("\nIndMin:\n");
    // for(unsigned int i=0;i<cntMin[0];i++) printf("%d ",indMin[i]);
    
    // Get all children
    unsigned int *resultIndex, *dresultIndex;
    unsigned int *cnt, *dcnt;
    resultIndex = (unsigned int *)malloc(cntMin[0]*sizeof(unsigned int));
    hipMalloc(&dresultIndex, cntMin[0]*sizeof(unsigned int));
    cnt = (unsigned int *)malloc(sizeof(unsigned int));
    hipMalloc(&dcnt, sizeof(unsigned int));

    int indexChild;
    for(int i=0;i<cntMin[0]-1;i++){
        num = (float)(i+1)/(float)BLOCK_SIZE;
        mod = BLOCK_SIZE;
        if(i+1 < BLOCK_SIZE) mod = (i+1)%BLOCK_SIZE;
        n = ceil(num);
        
        searchSimilarIndex<<<n, mod>>>(dindMin, dresultIndex, dcnt, i);
        hipDeviceSynchronize();

        hipMemcpy(resultIndex, dresultIndex, cntMin[0]*sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(cnt, dcnt, sizeof(unsigned int), hipMemcpyDeviceToHost);
        
        if(cnt[0] == 0) indexChild = indMin[i];
        else indexChild = *std::max_element(resultIndex, resultIndex + cnt[0])-1;
        buildHuffmanTree(cnt[0], uniqueChar, tempFreq, i, indexChild);

    }
    // for(int j=0;j<cntMin[0]-1;j++){
    //         printf("Index %d:Frequency %u",j,huffmanTreeNode[j].frequency);
    //         if(huffmanTreeNode[j].letter != '\0') printf(":Letter %c\n",huffmanTreeNode[j].letter);
    //         if(huffmanTreeNode[j].left != NULL) printf(":Left %u:Right %u\n",(huffmanTreeNode[j].left)->frequency,(huffmanTreeNode[j].right)->frequency);
    //     }

    
    // build the huffman dictionary with the bit sequence and its length
    buildHuffmanDictionary(huffmanNode_head, bitSequence, bitSequenceLength);

    // write data to the file
    outputData = (unsigned char *)malloc(outputFileLength * sizeof(unsigned char));
    huffmanNode_current = huffmanNode_head;
    outputFileLengthCounter = 0;
    for(int i = 0; i < compressedFileLength-extra; i++){

            // value is 0 then left sub tree
            if(compressedData[i] == 0){
                huffmanNode_current = huffmanNode_current -> left;
                if(huffmanNode_current -> left == NULL){
                    outputData[outputFileLengthCounter] = huffmanNode_current -> letter;
                    huffmanNode_current = huffmanNode_head;
                    outputFileLengthCounter ++;
                }
            }
            // value is 1 the right sub tree
            else {
                huffmanNode_current = huffmanNode_current -> right;
                if(huffmanNode_current -> right == NULL){
                    outputData[outputFileLengthCounter] = huffmanNode_current -> letter;
                    huffmanNode_current = huffmanNode_head;
                    outputFileLengthCounter ++;
                }
            }
    }

    // end the clock, tick tick
    end = clock();

    // write the data to the output file
    outputFile = fopen(argv[2], "wb");
    
    fwrite(outputData, sizeof(unsigned char), outputFileLength, outputFile);
    fclose(outputFile);

    // printing debug info if debug is on
    if(DEBUG){
        printf("\nCompressed file length :: %d", compressedFileLength/8);
        //printf("\nOutput file length counter :: %d", outputFileLengthCounter);
        printf("\nOutput file length :: %d", outputFileLength);
        // printf("\nMerged Huffman Nodes :: %d", mergedHuffmanNodes);
        // printf("\nDistinct character count :: %d", distinctCharacterCount);
    }

    cpuTime = (end - start) * 1000 / CLOCKS_PER_SEC;
    printf("\nTime taken: %d:%d s\n", cpuTime / 1000, cpuTime % 1000);

	// clean up
	free(outputData);
	free(compressedData);

	return 0;
}
